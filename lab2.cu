
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>


#define CSC(call) 														                        \
while (1) 																				        \
{																							    \
	hipError_t status = call;									                                \
	if (status != hipSuccess) {								                                \
		printf("ERROR in %s:%d. Message: %s\n", __FILE__, __LINE__, hipGetErrorString(status));\
		exit(0);																	            \
	}																						    \
	break;																					    \
}

__global__ void kernel(hipTextureObject_t tex, uchar4 *out, int w, int h)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
    int offset_x = blockDim.x * gridDim.x;
    int offset_y = blockDim.y * gridDim.y;
    int x, y;
    uchar4 p;
    for(y = idy; y < h; y += offset_y)
        for(x = idx; x < w; x += offset_x) {
            float Gx = 0.0;
            float Gy = 0.0;

            y = max(min(y, h), 0);
            x = max(min(x, w), 0);

            //horizontal
            for (int ky = -1; ky <= 1; ky++) {
                for (int kx = -1; kx <= 1; kx += 2) {
                    p = tex2D<uchar4>(tex, x + kx, y + ky);
                    float Y = 0.299 * p.x + 0.587 * p.y + 0.114 * p.z;
                    Gx += kx * Y;
                }
            }

            // vertical
            for (int ky = -1; ky <= 1; ky += 2) {
                for (int kx = -1; kx <= 1; kx++) {
                    p = tex2D<uchar4>(tex, x + kx, y + ky);
                    float Y = 0.299 * p.x + 0.587 * p.y + 0.114 * p.z;
                    Gy += ky * Y;
                }
            }
        
            float gradient = sqrt(Gx * Gx + Gy * Gy);
            gradient = min(max(gradient, 0.0f), 255.0f);

            out[y * w + x] = make_uchar4(gradient, gradient, gradient, p.w);
        }
}

int main() 
{
    char inputFile[255], outputFile[255];
    scanf("%s", inputFile);
    scanf("%s", outputFile);

    int width, height;
    FILE *fp = fopen(inputFile, "rb");
    fread(&width, sizeof(int), 1, fp);
    fread(&height, sizeof(int), 1, fp);
    uchar4 *data = (uchar4 *)malloc(sizeof(uchar4) * width * height);
    fread(data, sizeof(uchar4), width * height, fp);
    fclose(fp);

    hipArray *arr;
    hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();
    CSC(hipMallocArray(&arr, &ch, width, height));
    CSC(hipMemcpy2DToArray(arr, 0, 0, data, width * sizeof(uchar4), width * sizeof(uchar4), height, hipMemcpyHostToDevice));

    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = arr;

    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = false;

    hipTextureObject_t tex = 0;
    CSC(hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL));

    uchar4 *dev_out;
    CSC(hipMalloc(&dev_out, sizeof(uchar4) * width * height));

    kernel<<< dim3(16, 16), dim3(32, 32) >>>(tex, dev_out, width, height);
    CSC(hipGetLastError());

    CSC(hipMemcpy(data, dev_out, sizeof(uchar4) * width * height, hipMemcpyDeviceToHost));

    CSC(hipDestroyTextureObject(tex));
    CSC(hipFreeArray(arr));
    CSC(hipFree(dev_out));

    fp = fopen(outputFile, "wb");
    fwrite(&width, sizeof(int), 1, fp);
    fwrite(&height, sizeof(int), 1, fp);
    fwrite(data, sizeof(uchar4), width * height, fp);
    fclose(fp);

    free(data);
    return 0;
}